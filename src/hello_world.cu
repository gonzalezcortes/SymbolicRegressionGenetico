#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void helloWorldKernel(int* array, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        array[index] = index;
    }
}

extern "C" void run_hello_world_kernel(int* host_array, int N) {
    int* device_array;

    // Allocate device memory
    hipMalloc((void**)&device_array, N * sizeof(int));

    // Launch the kernel
    helloWorldKernel << <1, N >> > (device_array, N);

    // Copy the array back to the host
    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_array);
}
