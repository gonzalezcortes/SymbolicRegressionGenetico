#include "hip/hip_runtime.h"
#include <stdio.h>

// Define a kernel function.
__global__ void helloWorldKernel(int* array, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        array[index] = index;
    }
}

extern "C" void run_hello_cuda(int* host_array, int N) {
    int* device_array;

    // Allocate device memory
    hipMalloc((void**)&device_array, N * sizeof(int));

    // Launch the kernel
    helloWorldKernel << <1, N >> > (device_array, N);

    // Copy the array back to the host
    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_array);
}
