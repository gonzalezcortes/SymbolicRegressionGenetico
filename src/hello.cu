#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <vector>
#include <stdio.h>

__global__ void sayHello() {
    printf("Hello world from the GPU!\n");
}

void launchSayHello() {
    sayHello << <1, 1 >> > ();  // Launch kernel with 1 block and 1 thread
    hipDeviceSynchronize();  // Make sure the kernel has finished
}

PYBIND11_MODULE(kernel, m) {
    m.def("sayHello", &launchSayHello, "A function to say hello from the GPU");
}
