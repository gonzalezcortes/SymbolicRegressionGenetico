#include "hip/hip_runtime.h"
#include <stdio.h>

// Define a kernel function.
__global__ void helloWorldKernel(int* array, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        array[index] = index;
    }
}

int main() {
    int N = 16; // The size of our array
    int* host_array, * device_array;

    // Allocate host memory
    host_array = (int*)malloc(N * sizeof(int));

    // Allocate device memory
    hipMalloc((void**)&device_array, N * sizeof(int));

    // Launch the kernel
    helloWorldKernel << <1, N >> > (device_array, N);

    // Copy the array back to the host
    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print out the resulting array
    printf("Hello from CUDA!\nArray: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", host_array[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(device_array);

    // Free host memory
    free(host_array);

    return 0;
}
