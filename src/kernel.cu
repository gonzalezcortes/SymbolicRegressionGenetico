#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <vector>

namespace py = pybind11;

void square(std::vector<int> &data) {
    int *dev_data;
    int size = data.size();
  
    hipMalloc((void **)&dev_data, size * sizeof(int));
    hipMemcpy(dev_data, data.data(), size * sizeof(int), hipMemcpyHostToDevice);
  
    square_kernel<<<(size + 255) / 256, 256>>>(dev_data, size);
  
    hipMemcpy(data.data(), dev_data, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_data);
}

PYBIND11_MODULE(kernel, m) {
    m.def("square", &square, "Square each element of the array");
}
